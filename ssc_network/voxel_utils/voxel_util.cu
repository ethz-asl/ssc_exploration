#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void depth2Grid(double *  cam_info, double *  vox_info,  double * depth_data, double * vox_binary_GPU, double * depth2voxel_idx ){
  // Get camera information
  int frame_width = cam_info[0];
  //int frame_height = cam_info[1];
  double cam_K[9];
  for (int i = 0; i < 9; ++i)
    cam_K[i] = cam_info[i + 2];
  double cam_pose[16];
  for (int i = 0; i < 16; ++i)
    cam_pose[i] = cam_info[i + 11];

  // Get voxel volume parameters
  double vox_unit = vox_info[0];
  //double vox_margin = vox_info[1];
  int vox_size[3];
  for (int i = 0; i < 3; ++i)
    vox_size[i] = vox_info[i + 2];
  double vox_origin[3];
  for (int i = 0; i < 3; ++i)
    vox_origin[i] = vox_info[i + 5];


  // Get point in world coordinate
  int pixel_x = blockIdx.x;
  int pixel_y = threadIdx.x;

  double point_depth = depth_data[pixel_y * frame_width + pixel_x];

  double point_cam[3] = {0};
  point_cam[0] =  (pixel_x - cam_K[2])*point_depth/cam_K[0];
  point_cam[1] =  (pixel_y - cam_K[5])*point_depth/cam_K[4];
  point_cam[2] =  point_depth;

  double point_base[3] = {0};

  point_base[0] = cam_pose[0 * 4 + 0]* point_cam[0] + cam_pose[0 * 4 + 1]*  point_cam[1] + cam_pose[0 * 4 + 2]* point_cam[2];
  point_base[1] = cam_pose[1 * 4 + 0]* point_cam[0] + cam_pose[1 * 4 + 1]*  point_cam[1] + cam_pose[1 * 4 + 2]* point_cam[2];
  point_base[2] = cam_pose[2 * 4 + 0]* point_cam[0] + cam_pose[2 * 4 + 1]*  point_cam[1] + cam_pose[2 * 4 + 2]* point_cam[2];

  point_base[0] = point_base[0] + cam_pose[0 * 4 + 3];
  point_base[1] = point_base[1] + cam_pose[1 * 4 + 3];
  point_base[2] = point_base[2] + cam_pose[2 * 4 + 3];


  //printf("vox_origin: %f,%f,%f\n",vox_origin[0],vox_origin[1],vox_origin[2]);
  // World coordinate to grid coordinate
  int z = (int)floor((point_base[0] - vox_origin[0])/vox_unit);
  int x = (int)floor((point_base[1] - vox_origin[1])/vox_unit);
  int y = (int)floor((point_base[2] - vox_origin[2])/vox_unit);
	
	//printf("calculating depth mappings");
  // mark vox_binary_GPU
  if( x >= 0 && x < vox_size[0] && y >= 0 && y < vox_size[1] && z >= 0 && z < vox_size[2]){
    int vox_idx = z * vox_size[0] * vox_size[1] + y * vox_size[0] + x;
    vox_binary_GPU[vox_idx] = double(1.0);
    //printf("depth mapping at %d,%d,%d is %d\n", x,y,z,vox_idx);
    depth2voxel_idx[pixel_y * frame_width + pixel_x] = vox_idx;
  }
}

__global__ void calculate_occupancy_prob(double *  cam_info, double *  vox_info,  double * depth_data, double * vox_log_GPU ){
  // Get camera information
  int frame_width = cam_info[0];
  //int frame_height = cam_info[1];
  double cam_K[9];
  for (int i = 0; i < 9; ++i)
    cam_K[i] = cam_info[i + 2];
  double cam_pose[16];
  for (int i = 0; i < 16; ++i)
    cam_pose[i] = cam_info[i + 11];

  // Get voxel volume parameters
  double vox_unit = vox_info[0];
  //double vox_margin = vox_info[1];
  int vox_size[3];
  for (int i = 0; i < 3; ++i)
    vox_size[i] = vox_info[i + 2];
  double vox_origin[3];
  for (int i = 0; i < 3; ++i)
    vox_origin[i] = vox_info[i + 5];


  // Get point in world coordinate
  int pixel_x = blockIdx.x;
  int pixel_y = threadIdx.x;

  double point_depth = depth_data[pixel_y * frame_width + pixel_x];

  double point_cam[3] = {0};
  point_cam[0] =  (pixel_x - cam_K[2])*point_depth/cam_K[0];
  point_cam[1] =  (pixel_y - cam_K[5])*point_depth/cam_K[4];
  point_cam[2] =  point_depth;

  double point_base[3] = {0};

  point_base[0] = cam_pose[0 * 4 + 0]* point_cam[0] + cam_pose[0 * 4 + 1]*  point_cam[1] + cam_pose[0 * 4 + 2]* point_cam[2];
  point_base[1] = cam_pose[1 * 4 + 0]* point_cam[0] + cam_pose[1 * 4 + 1]*  point_cam[1] + cam_pose[1 * 4 + 2]* point_cam[2];
  point_base[2] = cam_pose[2 * 4 + 0]* point_cam[0] + cam_pose[2 * 4 + 1]*  point_cam[1] + cam_pose[2 * 4 + 2]* point_cam[2];

  point_base[0] = point_base[0] + cam_pose[0 * 4 + 3];
  point_base[1] = point_base[1] + cam_pose[1 * 4 + 3];
  point_base[2] = point_base[2] + cam_pose[2 * 4 + 3];


  //printf("vox_origin: %f,%f,%f\n",vox_origin[0],vox_origin[1],vox_origin[2]);
  // World coordinate to grid coordinate
  int z = (int)floor((point_base[0] - vox_origin[0])/vox_unit);
  int x = (int)floor((point_base[1] - vox_origin[1])/vox_unit);
  int y = (int)floor((point_base[2] - vox_origin[2])/vox_unit);
	
	//printf("calculating depth mappings");
  // mark vox_binary_GPU
  float prob_occ = 0.7f;
  float max_prob = 0.97f;
  
  if( x >= 0 && x < vox_size[0] && y >= 0 && y < vox_size[1] && z >= 0 && z < vox_size[2]){
    int vox_idx = z * vox_size[0] * vox_size[1] + y * vox_size[0] + x;
    vox_log_GPU[vox_idx] = min(vox_log_GPU[vox_idx] + log(prob_occ/(1-prob_occ)), log(max_prob/(1-max_prob)));
  }
}

__global__
void tsdfTransform( double * vox_info, double * vox_tsdf){

  int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (vox_idx >= vox_info[0+2] * vox_info[1+2] * vox_info[2+2]){
    return;
  }
  double value = double(vox_tsdf[vox_idx]);


  double sign;
  if (abs(value) < 0.001)
    sign = 1;
  else
    sign = value/abs(value);

  vox_tsdf[vox_idx] = sign*(max(0.001,(1.0-abs(value))));
}


__global__ void SquaredDistanceTransform(double * cam_info, double * vox_info, double * depth_data, double * vox_binary_GPU , double * vox_tsdf) {
  // Get voxel volume parameters
  double vox_unit = vox_info[0];
  double vox_margin = vox_info[1];
  int vox_size[3];
  for (int i = 0; i < 3; ++i)
    vox_size[i] = vox_info[i + 2];
  double vox_origin[3];
  for (int i = 0; i < 3; ++i)
    vox_origin[i] = vox_info[i + 5];

  int frame_width = cam_info[0];
  int frame_height = cam_info[1];
  double cam_K[9];
  for (int i = 0; i < 9; ++i)
    cam_K[i] = cam_info[i + 2];
  double cam_pose[16];
  for (int i = 0; i < 16; ++i)
    cam_pose[i] = cam_info[i + 11];


  int vox_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (vox_idx >= vox_size[0] * vox_size[1] * vox_size[2]){
    return;
  }

  int z = double((vox_idx / ( vox_size[0] * vox_size[1]))%vox_size[2]) ;
  int y = double((vox_idx / vox_size[0]) % vox_size[1]);
  int x = double(vox_idx % vox_size[0]);
  int search_region = (int)round(vox_margin/vox_unit);

  if (vox_binary_GPU[vox_idx] >0 ){
    vox_tsdf[vox_idx] = 0;
    return;
  }

  // Get point in world coordinates (XYZ) from grid coordinates (YZX)  
  double point_base[3] = {0};
  point_base[0] = double(z) * vox_unit + vox_origin[0];
  point_base[1] = double(x) * vox_unit + vox_origin[1];
  point_base[2] = double(y) * vox_unit + vox_origin[2];

  // Get point in current camera coordinates
  double point_cam[3] = {0};
  point_base[0] = point_base[0] - cam_pose[0 * 4 + 3];
  point_base[1] = point_base[1] - cam_pose[1 * 4 + 3];
  point_base[2] = point_base[2] - cam_pose[2 * 4 + 3];
  point_cam[0] = cam_pose[0 * 4 + 0] * point_base[0] + cam_pose[1 * 4 + 0] * point_base[1] + cam_pose[2 * 4 + 0] * point_base[2];
  point_cam[1] = cam_pose[0 * 4 + 1] * point_base[0] + cam_pose[1 * 4 + 1] * point_base[1] + cam_pose[2 * 4 + 1] * point_base[2];
  point_cam[2] = cam_pose[0 * 4 + 2] * point_base[0] + cam_pose[1 * 4 + 2] * point_base[1] + cam_pose[2 * 4 + 2] * point_base[2];
  if (point_cam[2] <= 0){
    return;
  }

  // Project point to 2D
  int pixel_x = roundf(cam_K[0] * (point_cam[0] / point_cam[2]) + cam_K[2]);
  int pixel_y = roundf(cam_K[4] * (point_cam[1] / point_cam[2]) + cam_K[5]);
  if (pixel_x < 0 || pixel_x >= frame_width || pixel_y < 0 || pixel_y >= frame_height){ // outside FOV
    return;
  }


  // Get depth
  double point_depth = depth_data[pixel_y * frame_width + pixel_x];
  if (point_depth < double(0.5f) || point_depth > double(8.0f)){
    return;
  }
  if (roundf(point_depth) == 0){ // mising depth
    vox_tsdf[vox_idx] = double(-1.0);
    return;
  }


  // Get depth difference
  double sign;
  if (abs(point_depth - point_cam[2]) < 0.0001){
    sign = 1; // avoid NaN
  }else{
    sign = (point_depth - point_cam[2])/abs(point_depth - point_cam[2]);
  }
  vox_tsdf[vox_idx] = double(sign);
  for (int iix = max(0,x-search_region); iix < min((int)vox_size[0],x+search_region+1); iix++){
    for (int iiy = max(0,y-search_region); iiy < min((int)vox_size[1],y+search_region+1); iiy++){
      for (int iiz = max(0,z-search_region); iiz < min((int)vox_size[2],z+search_region+1); iiz++){
        int iidx = iiz * vox_size[0] * vox_size[1] + iiy * vox_size[0] + iix;
        if (vox_binary_GPU[iidx] > 0){
          double xd = abs(x - iix);
          double yd = abs(y - iiy);
          double zd = abs(z - iiz);
          double tsdf_value = sqrtf(xd * xd + yd * yd + zd * zd)/(double)search_region;
          if (tsdf_value < abs(vox_tsdf[vox_idx])){
            vox_tsdf[vox_idx] = double(tsdf_value*sign);
          }
        }
      }
    }
  }

}

void ComputeTSDF(double * cam_info_CPU, double * vox_info_CPU,
                 double * depth_data_CPU,  double * vox_tsdf_CPU, double * depth_mapping_idxs_CPU, double * occupancy) {

  int frame_width  = cam_info_CPU[0];
  int frame_height = cam_info_CPU[1];
  int vox_size[3];
  for (int i = 0; i < 3; ++i)
    vox_size[i] = vox_info_CPU[i + 2];
  int num_crop_voxels = vox_size[0] * vox_size[1] * vox_size[2];


  // allocate voxel occupancy
  double * vox_binary_CPU = (double*)malloc((int)(num_crop_voxels * sizeof(double)));
	memset(vox_binary_CPU, 0, num_crop_voxels * sizeof(double));

  //  Copy from host to device
  double *  vox_binary_GPU;
  hipMalloc(&vox_binary_GPU, num_crop_voxels * sizeof(double));
  hipMemcpy(vox_binary_GPU, vox_binary_CPU, num_crop_voxels * sizeof(double), hipMemcpyHostToDevice);
  //GPU_set_zeros(num_crop_voxels, vox_binary_GPU);

  // copy cam info to gpu
  double * cam_info_GPU;
  hipMalloc(&cam_info_GPU, 27 * sizeof(double));
  hipMemcpy(cam_info_GPU, cam_info_CPU, 27 * sizeof(double), hipMemcpyHostToDevice);

  // copy vox info to gpu
  double * vox_info_GPU;
  hipMalloc(&vox_info_GPU, 8 * sizeof(double));
  hipMemcpy(vox_info_GPU, vox_info_CPU, 8 * sizeof(double), hipMemcpyHostToDevice);

  //copy depth data to gpu
  double * depth_data_GPU;
  hipMalloc(&depth_data_GPU, frame_height * frame_width * sizeof(double));
  hipMemcpy(depth_data_GPU, depth_data_CPU, frame_height * frame_width * sizeof(double), hipMemcpyHostToDevice);

  // copy depth mapping to gpu
  double * depth_mapping_idxs_GPU;
  hipMalloc(&depth_mapping_idxs_GPU, frame_height * frame_width * sizeof(double));
  hipMemcpy(depth_mapping_idxs_GPU, depth_mapping_idxs_CPU, frame_height * frame_width * sizeof(double), hipMemcpyHostToDevice);

  // copy voxel tsd to gpu
  double * vox_tsdf_GPU;
  hipMalloc(&vox_tsdf_GPU, num_crop_voxels * sizeof(double));
  hipMemcpy(vox_tsdf_GPU, vox_tsdf_CPU, num_crop_voxels * sizeof(double), hipMemcpyHostToDevice);

  // from depth map to binaray voxel representation 
  depth2Grid<<<frame_width,frame_height>>>(cam_info_GPU, vox_info_GPU, depth_data_GPU, vox_binary_GPU, depth_mapping_idxs_GPU);

  //cudaGetLastError();

  // distance transform 
  int THREADS_NUM = 512;  // 1024
  int BLOCK_NUM = int((num_crop_voxels + size_t(THREADS_NUM) - 1) / THREADS_NUM);

  SquaredDistanceTransform <<< BLOCK_NUM, THREADS_NUM >>> (cam_info_GPU, vox_info_GPU, depth_data_GPU, vox_binary_GPU, vox_tsdf_GPU);
  
  // invert TSDF
  tsdfTransform <<< BLOCK_NUM, THREADS_NUM >>> (vox_info_GPU, vox_tsdf_GPU);
  
  // copy computed TSDF back to CPU
  hipMemcpy(vox_tsdf_CPU, vox_tsdf_GPU, num_crop_voxels * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(depth_mapping_idxs_CPU, depth_mapping_idxs_GPU, frame_height * frame_width * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(occupancy, vox_binary_GPU,  num_crop_voxels * sizeof(double), hipMemcpyDeviceToHost);
  
  // deallocation
  hipFree(vox_info_GPU);
  hipFree(cam_info_GPU);
  hipFree(depth_data_GPU);
  hipFree(depth_mapping_idxs_GPU);
  hipFree(vox_tsdf_GPU);
  hipFree(vox_binary_GPU);
  free(vox_binary_CPU);
}

void calculateOccupancyProb(double * cam_info_CPU, double * vox_info_CPU,
                 double * depth_data_CPU, double * log_odds_occupancy) {

  int frame_width  = cam_info_CPU[0];
  int frame_height = cam_info_CPU[1];
  int vox_size[3];
  for (int i = 0; i < 3; ++i)
    vox_size[i] = vox_info_CPU[i + 2];
  int num_crop_voxels = vox_size[0] * vox_size[1] * vox_size[2];


  // allocate voxel occupancy
  double * vox_prob_CPU = (double*)malloc((int)(num_crop_voxels * sizeof(double)));
	memset(vox_prob_CPU, 0, num_crop_voxels * sizeof(double));

  //  Copy from host to device
  double *  vox_prob_GPU;
  hipMalloc(&vox_prob_GPU, num_crop_voxels * sizeof(double));
  hipMemcpy(vox_prob_GPU, vox_prob_CPU, num_crop_voxels * sizeof(double), hipMemcpyHostToDevice);
  //GPU_set_zeros(num_crop_voxels, vox_binary_GPU);

  // copy cam info to gpu
  double * cam_info_GPU;
  hipMalloc(&cam_info_GPU, 27 * sizeof(double));
  hipMemcpy(cam_info_GPU, cam_info_CPU, 27 * sizeof(double), hipMemcpyHostToDevice);

  // copy vox info to gpu
  double * vox_info_GPU;
  hipMalloc(&vox_info_GPU, 8 * sizeof(double));
  hipMemcpy(vox_info_GPU, vox_info_CPU, 8 * sizeof(double), hipMemcpyHostToDevice);

  //copy depth data to gpu
  double * depth_data_GPU;
  hipMalloc(&depth_data_GPU, frame_height * frame_width * sizeof(double));
  hipMemcpy(depth_data_GPU, depth_data_CPU, frame_height * frame_width * sizeof(double), hipMemcpyHostToDevice);


  // from depth map to binary voxel representation 
  calculate_occupancy_prob<<<frame_width,frame_height>>>(cam_info_GPU, vox_info_GPU, depth_data_GPU, vox_prob_GPU);

  // copy computed log odds back to CPU
  hipMemcpy(log_odds_occupancy, vox_prob_GPU,  num_crop_voxels * sizeof(double), hipMemcpyDeviceToHost);
  
  // deallocation
  hipFree(vox_info_GPU);
  hipFree(cam_info_GPU);
  hipFree(depth_data_GPU);
  hipFree(vox_prob_GPU);
  free(vox_prob_CPU);
}
//int main() {return 0;}
